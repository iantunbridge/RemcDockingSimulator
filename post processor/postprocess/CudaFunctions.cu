#include "hip/hip_runtime.h"
// Replica Kernel functions to call cuda kernels for energy calculations in a replica
#ifndef CUDA_FUNCTIONS_CU
#define CUDA_FUNCTIONS_CU

using namespace std;

#include "CudaFunctions.h"
#include <cutil.h>
#include <cutil_inline.h>
// Block index
#define bx blockIdx.x
#define by blockIdx.y

// Thread index
#define tx threadIdx.x
#define ty threadIdx.y

#if USING_CUDA

#if LJ_LOOKUP_METHOD == TEXTURE_MEM

void bindLJTexture(float * ljp)
{
	size_t offset;
	hipBindTexture(&offset, LJTexture, ljp, sizeof(float)*AA_COUNT*AA_COUNT);
	if (int(offset)>0)
		printf("!!! Texture offset %d\n",int(offset));
}

void unbindLJTexture()
{
	hipUnbindTexture(LJTexture);
}

/*void bindLJTexture2D(float *ljp)
{
	const textureReference *texRefPtr;
	hipGetTextureReference(&texRefPtr, HIP_SYMBOL("LJTexture2D"));
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	LJTexture2D.addressMode[0] = hipAddressModeWrap;
	LJTexture2D.addressMode[1] = hipAddressModeWrap;
	LJTexture2D.filterMode = hipFilterModePoint;
	LJTexture2D.normalized = false;

	size_t offset;
	hipBindTexture2D(&offset, texRefPtr, ljp, &channelDesc, AA_COUNT, AA_COUNT, AA_COUNT*sizeof(float)+1);

	if (int(offset)>0)
		printf("!!! Texture offset %d\n",int(offset));
}*/


/*void unbindLJTexture2D ()
{
	const textureReference *texRefPtr;
	hipGetTextureReference(&texRefPtr, HIP_SYMBOL("LJTexture2D"));
	hipUnbindTexture(texRefPtr);
}*/

#endif

#if METADATA_MEMORY == TEXTURE_MEM
int bindMetaDataToTexture(void* deviceMemory, size_t size)
{
	return hipBindTexture(0, residueMetaTex, deviceMemory, size);
}

int freeMetaDataTexture()
{
	return hipUnbindTexture(residueMetaTex);
}
#endif

#if POSITIONDATA_MEMORY == TEXTURE_MEM
int bindPositionDataToTexture(void* deviceMemory, size_t size)
{
	return hipBindTexture(0, residuePositionTex, deviceMemory, size);
}

int freePositionDataTexture()
{
	return hipUnbindTexture(residuePositionTex);
}
#endif

float * LJPotentialDataToDevice (AminoAcids *a)
{
	float * _LJPotentialData;
	long tableSize(sizeof(float[AA_COUNT*AA_COUNT]));
	long containerSize(tableSize);  // pad the container such that shared memory tables align nicely when preloading

	#if LJ_LOOKUP_METHOD == SHARED_MEM
		containerSize = TILE_DIM * ceil(float(tableSize)/float(TILE_DIM));
	#endif

	float * safeData = new float[AA_COUNT*AA_COUNT];

	for (int i=0;i<AA_COUNT;i++)
	{
		memcpy (&safeData[i*AA_COUNT],&a->LJpotentials[i][0],sizeof(float)*AA_COUNT);
	}

	CUDA_SAFE_CALL(hipMalloc((void**)&_LJPotentialData,containerSize));
	CUDA_SAFE_CALL(hipMemset(_LJPotentialData,0,containerSize));
	// copy host memory to device
	CUDA_SAFE_CALL(hipMemcpy(_LJPotentialData, safeData, tableSize, hipMemcpyHostToDevice) );
	delete [] safeData;
   	return _LJPotentialData;
}

void copyLJPotentialDataToDevice (float * dev_LJPotentialData, AminoAcids *a)
{
	#define mem_size int(sizeof(float)*AA_COUNT*AA_COUNT)
	#define table_size (AA_COUNT*AA_COUNT)
	float * safeData = new float[table_size];

	for (int i=0;i<table_size;i++)
		safeData[i] = 0.0f;

	for (int i=0;i<AA_COUNT;i++)
	{
		memcpy (&safeData[i*AA_COUNT],a->LJpotentials[i],sizeof(float)*AA_COUNT);
	}

	//CUDA_SAFE_CALL(hipMalloc((void**)&dev_LJPotentialData,mem_size));
	// copy host memory to device
	hipMemcpy(dev_LJPotentialData, safeData, mem_size, hipMemcpyHostToDevice);
	cutilCheckMsg("Failed to copy contact potentials");
	delete [] safeData;
}

void cudaInfo()
{
	int deviceCount;
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0)
	{
		printf("ERROR: no devices supporting CUDA.\n");
	    exit(EXIT_FAILURE);
	}
	#if OUTPUT_LEVEL > 0
	struct hipDeviceProp_t devInfo;
	printf("\n------------------------------------\n");
	for (int i=0;i<deviceCount;i++)
	{
		hipGetDeviceProperties( &devInfo, 0 );
		printf ("CUDA Device %d Info:\n",i);
		printf("Name: %s\n", devInfo.name );
		printf("totalGlobalMem: %i\n",(int)devInfo.totalGlobalMem );
		printf("sharedMemPerBlock: %i\n",(int)devInfo.sharedMemPerBlock );
		printf("regsPerBlock: %i\n",devInfo.regsPerBlock );
		printf("warpSize: %i\n",devInfo.warpSize );
		printf("memPitch %i\n",(int)devInfo.memPitch );
		printf("maxThreadsPerBlock: %i\n",devInfo.maxThreadsPerBlock);
		if (i < deviceCount-1)
			printf("\n");
	}
	printf("------------------------------------\n");
	#endif
}

void MCSearchOnDevice()
{
	printf("MCSearchOnDevice() does nothing\n");
};

// stream kernel for overlapping calls
void CUDA_EonDevice_async(float4 *residuePositions, float4 *residueMeta, int * residueCount, int *moleculePositions, int *moleculeCount, float* LJPotentials, float* kernelResult, int resultSize, int blockSize, int datasetSize, int sm_size, hipStream_t stream)
{
	dim3 dimBlock(blockSize,1,1);   // threads
	dim3 dimGrid(datasetSize/blockSize,datasetSize/blockSize,1);    	// blocks

	// Launch the device computation
	// result must become an array of dimentions the same as grid so that the multiple thread blocks can sum in it properly
	//hipStreamSynchronize(stream);

	E_TiledKernel<<< dimGrid,dimBlock,sm_size,stream >>>(residuePositions, residueMeta, residueCount, moleculePositions, moleculeCount, LJPotentials, kernelResult);

	cutilCheckMsg("Kernel execution failed");
	//hipMemcpyAsync(hostResult,kernelResult, sizeof(float)*resultSize*resultSize, hipMemcpyDeviceToHost,stream);
	return;
};

void CUDA_Esum_async(float* result, float *d_resultMatrix, int resultSize, int datasetSize, hipStream_t stream)
{
	// parallel sum on gpu if required
}
/*
void CUDA_EonDevice_B(float4 *residuePositions, float4 *residueMeta, int * residueCount, int *moleculePositions, int *moleculeCount, float* LJPotentials, float* result, int blockSize, int datasetSize)
{
	// result stored on the device
	// gridSize can be arbitrary
	int gridSize = datasetSize/blockSize;
	// the parallel sum of the results must have an array with size that is a power of 2
	int resultSize = int(pow(2,ceil(log(float(gridSize))/log(2.0))));
	//resultSize *= resultSize;

	//for a parallel sum each grid must have one cell in the array of results from all the threads
	float *d_result;
	hipMalloc((void **)&d_result,sizeof(float)*resultSize*resultSize);
	hipMemset(d_result,0,sizeof(float)*resultSize*resultSize);


	// Launch the device computation
	// result must become an array of dimentions the same as grid so that the multiple thread blocks can sum in it properly

	dim3 dimBlock(blockSize,blockSize,1);   // threads
	dim3 dimGrid(gridSize,gridSize,1);    	// blocks

	E_SimpleKernel<<< dimGrid,dimBlock >>>(residuePositions, residueMeta, residueCount, moleculePositions, moleculeCount, LJPotentials, d_result);


	//sum the results from the thread blocks on the GPU
	 fix this for multiple reduction sums!!!

	//    float * sumResults;
	//
	//    if (resultSize*resultSize<=512) //options are 1 ... 256 == 16x16 thread blocks, ie fits in one kernel
	//    {
	//    	parallelSum_kernel<<<1,resultSize*resultSize>>>(d_result,d_stride);  // THIS WORKS!
	//	}
	//    else   // resultsize > 512 == 32x32,64x64,128x128,256x256 .... etc
	//    {
	//    	int threadBlockSize = 512;				// maximum coverage by threads
	//    	gridSize = resultSize*resultSize/512;   // how many grids we need
	//	   	hipMalloc((void **)&sumResults,sizeof(float)*gridSize);
	//		hipMemset(sumResults,0,sizeof(float)*gridSize);
	//	   	parallelSum_kernel<<<gridSize,threadBlockSize>>>(d_result,sumResults);
	//
	//	}
	//	hipFree(d_stride);
	//	hipMemcpy(result,d_result, sizeof(float), hipMemcpyDeviceToHost);

	#ifdef _EMU

	printf("After || sum\n");
	for (int i=0;i<resultSize;i++)
	{
		for (int j=0;j<resultSize;j++)
			printf (" %10.7f",d_result[i*resultSize+j]);
		printf("\n");
	}
	#endif
	// copy the result back to host memory
	// sum it on the CPU, seems faster than a kernel for 64x64 at least

		float *tmpSums = new float [resultSize*resultSize];
		hipMemcpy(tmpSums,d_result, sizeof(float)*resultSize*resultSize, hipMemcpyDeviceToHost);
		result[0] = 0;
		for (int i=0;i<resultSize*resultSize;i++)
			result[0] += tmpSums[i];
		result[0] *= KBTConversionFactor;

	// free the memory assigned for this iteration.
	hipFree(d_result);

	return;
};


void CUDA_rotateMolecule (float4 *d_residuePositions, int *d_startPosition, int *d_moleculeLength, int moleculeLength, float4* d_rotationVector, float4* d_center, hipStream_t stream)
{
	int grids = int(ceil(float(moleculeLength)/512.0f));
	int threads = int(ceil(float(moleculeLength)/float(grids)));
	dim3 dimBlock(moleculeLength,1,1);   // threads
	dim3 dimGrid(grids,1,1);    	// blocks

	rotateMolecule_kernel<<<dimGrid,dimBlock,0,stream>>>(d_residuePositions, d_startPosition, d_moleculeLength, d_rotationVector, d_center);
}
*/
/*
void CUDA_translateMolecule (float4 *d_residuePositions, int *d_startPosition, int *d_moleculeLength, int moleculeLength, float4* d_translation, float4* d_center, hipStream_t stream)
{
	int grids = int(ceil(float(moleculeLength)/512.0f));
	int threads = int(ceil(float(moleculeLength)/float(grids)));
	dim3 dimBlock(moleculeLength,1,1);   // threads
	dim3 dimGrid(grids,1,1);    	// blocks

	translateMolecule_kernel <<< dimGrid,dimBlock,0,stream >>> (d_residuePositions, d_startPosition, d_moleculeLength, d_translation, d_center);
}

*/
// alternative summation algorithm will work with up to 33554432 residues
void CUDA_EonDevice(float4 *residuePositions, float4 *residueMeta, int * residueCount, int *moleculePositions, int *moleculeCount, float* LJPotentials, double* result, int blockSize, int datasetSize, int sm_size)
{

	// result stored on the device
	// gridSize can be arbitrary
	int gridSize = datasetSize/blockSize;
	// the parallel sum of the results must have an array with size that is a power of 2
	int resultSize = gridSize; //int(pow(2,ceil(log(float(gridSize))/log(2.0))));
	float *tmpSums = new float[resultSize*resultSize];

	//for a parallel sum each grid must have one cell in the array of results from all the threads
	float *d_result;
	hipMalloc((void **)&d_result,sizeof(float)*resultSize*resultSize);
	hipMemset(d_result,0,sizeof(float)*resultSize*resultSize);

	//float4 testmem;
	//hipMemcpy(&testmem,residuePositions, sizeof(float4), hipMemcpyDeviceToHost);


	// Launch the device computation
	// result must become an array of dimensions the same as grid so that the multiple thread blocks can sum in it properly

	dim3 dimBlock(blockSize,1,1);   		// threads
	dim3 dimGrid(gridSize,gridSize,1);    	// blocks


	E_TiledKernel<<< dimGrid,dimBlock,sm_size,0 >>>(residuePositions, residueMeta, residueCount, moleculePositions, moleculeCount, LJPotentials, d_result);
    	// check if kernel invocation generated an error

#ifdef _EMU

	printf("After kernel:\n");
	for (int i=0;i<resultSize;i++)
	{
		for (int j=0;j<resultSize;j++)
			printf (" %10.6f",d_result[i*resultSize+j]);
		printf("\n");
	}
#endif

#if PARALLEL_SUM
	parallelSum_kernel<<<1,gridSize>>>(d_result);
	hipMemcpy(result,d_result, sizeof(float), hipMemcpyDeviceToHost);
	result[0] *= KBTConversionFactor;
#else
	hipMemcpy(tmpSums,d_result, sizeof(float)*resultSize*resultSize, hipMemcpyDeviceToHost);
	result[0] = 0.0f;
	float c(0.0f);
	/*
			function kahanSum(input)
			 var potential = 0
			 var c = 0
			 for i = 0 to blockdim-1
			  y = lj+dh - c
			  t = potential + y
			  c = (t - potential) - y
			  potential = t
			 next i
			return sum
			*/

	for (int i=0;i<resultSize*resultSize;i++)
	{
		#if COMPENSATE_KERNEL_SUM
		float y(tmpSums[i] -c);
		float t((*result) + y);
		c = (t-(*result)) -y;
		*result = t;
		#else
		result[0] += tmpSums[i];
		#endif
	}
	result[0] *= KBTConversionFactor;

	delete [] tmpSums;
#endif
	// free the memory assigned for this iteration.
	hipFree(d_result);
	cutilCheckMsg("Kernel execution failed");
	return;
};

void CUDA_EonDeviceTest(float *d_x, float *d_y,float *d_z, int *d_id, float4 *residueMeta, float* LJPotentials, float* result, int blockSize, int datasetSize)
{

	// result stored on the device
	// gridSize can be arbitrary
	int gridSize(datasetSize/blockSize);
	float *tmpSums = new float[gridSize*gridSize];

	//for a parallel sum each grid must have one cell in the array of results from all the threads
	float *d_result;
	hipMalloc((void **)&d_result,sizeof(float)*gridSize*gridSize);
	hipMemset(d_result,0,sizeof(float)*gridSize*gridSize);

	//float4 testmem;
	//hipMemcpy(&testmem,residuePositions, sizeof(float4), hipMemcpyDeviceToHost);


	// Launch the device computation
	// result must become an array of dimensions the same as grid so that the multiple thread blocks can sum in it properly

	dim3 dimBlock(blockSize,1,1);   		// threads
	dim3 dimGrid(gridSize,gridSize,1);    	// blocks

	printf ("+++++++++++++++++++ NO KERNEL EXECUTED +++++++++++++++++++++++++++++");
	//E_TestTiledKernel<<< dimGrid,dimBlock >>>(d_x,d_y,d_z,d_id, residueMeta, LJPotentials, d_result);

	hipMemcpy(tmpSums,d_result, sizeof(float)*gridSize*gridSize, hipMemcpyDeviceToHost);
	result[0] = 0.0f;
	for (int i=0;i<gridSize*gridSize;i++)
	{
		result[0] += tmpSums[i];
	}
	result[0] *= KBTConversionFactor;
	delete [] tmpSums;

	// free the memory assigned for this iteration.
	hipFree(d_result);
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

	return;
};

/*void CUDA_EonDeviceLoop(float4 *residuePositions, float4 *residueMeta, int * residueCount, int *moleculePositions, int *moleculeCount, float* LJPotentials, float* result, int blockSize, int datasetSize)
{
	// result stored on the device
	// gridSize can be arbitrary
	int gridSize = datasetSize/blockSize;
	// the parallel sum of the results must have an array with size that is a power of 2
	int resultSize = gridSize; //int(pow(2,ceil(log(float(gridSize))/log(2.0))));

	//for a parallel sum each grid must have one cell in the array of results from all the threads
	float *d_result;
	hipMalloc((void **)&d_result,sizeof(float)*resultSize);
	hipMemset(d_result,0,sizeof(float)*resultSize);

	// Launch the device computation
	// result must become an array of dimensions the same as grid so that the multiple thread blocks can sum in it properly
	dim3 dimBlock(blockSize,1,1);   		// threads
	dim3 dimGrid(gridSize,1,1);    	// blocks



#ifdef _EMU

	printf("After kernel:\n");
	for (int i=0;i<resultSize;i++)
	{
		for (int j=0;j<resultSize;j++)
			printf (" %10.6f",d_result[i*resultSize+j]);
		printf("\n");
	}
#endif

#if PARALLEL_SUM
	parallelSum_kernel<<<1,gridSize>>>(d_result);
	hipMemcpy(result,d_result, sizeof(float), hipMemcpyDeviceToHost);
	result[0] *= KBTConversionFactor;
#else
	float *tmpSums = new float[resultSize];
	hipMemcpy(tmpSums,d_result, sizeof(float)*resultSize, hipMemcpyDeviceToHost);
	result[0] = 0.0f;
	for (int i=0;i<resultSize;i++)
	{
		result[0] += tmpSums[i];
	}
	result[0] *= KBTConversionFactor;
	delete [] tmpSums;
#endif
	// free the memory assigned for this iteration.
	hipFree(d_result);
	return;
};


__global__ void MCSearchOnDevice_kernel()
{
	return;
};
*/

__device__ float length(const float4 a, const float4 b)
{
	return sqrtf((a.x-b.x)*(a.x-b.x)+(a.y-b.y)*(a.y-b.y)+(a.z-b.z)*(a.z-b.z));

};

__device__ float crowderPotential(float r)
{
	return __powf(6.0f/r,6.0f);
};

__device__ float dhPotential(float q1, float q2, float r)
{
	return  q1 * q2 *__expf(-r/Xi) / r ;
};
/*
__global__ void E_SimpleKernel(float4 * residuePositions, float4 * residueMeta, int * residueCount, int * moleculePositions, int * moleculeCount, float* LJPotentialData, float* result)
{
	#define	 residueIp residuePositions[bx*blockDim.x+tx]
	#define residueIm residueMeta[bx*blockDim.x+tx]

	#define	 residueJp residuePositions[by*blockDim.x+ty]
	#define residueJm residueMeta[by*blockDim.x+ty]

	__shared__ float sharedmem_results[SHARED_MEM_RESULT_SIZE];
			// array of temp results to sum, can never be > 512
			// because of the number of threads allowed on a gpu
			// and is the only fixed memoy requirement I have in
			// this code so its set at 512


	// set each tmp_result to 0 if we are going to be using it.
	// problems here are that if the blockdim is not a power of 2 or too small, not all the
	// required cells are set to 0, resulting in an incorrect summation. the second line fixes
	// this provided SHARED_MEM_RESULT_SIZE is eq the next power of 2 up from the blocksize
	// squared. eg: BLOCK_SIZE == 12 then 2^ ceil((log(12)/log(2)) == 16 => SHARED_MEM_RESULT_SIZE == 256
	sharedmem_results[tx*blockDim.x+ty] = 0.0f;
	sharedmem_results[SHARED_MEM_RESULT_SIZE-(tx*blockDim.x-ty)] = 0.0f;

	#if LJ_LOOKUP_METHOD == SHARED_MEM_LJ
	// copy the LJ lookup table into shared memory
	__shared__ float sharedLJ[400];
	for (int i=0;i< ceil(400.0f/(blockDim.x*blockDim.y)) ;i++)
		sharedLJ[tx*blockDim.x+ty +blockDim.x*blockDim.y*i] = LJPotentialData[tx*blockDim.x+ty +blockDim.x*blockDim.y*i]; // basically there are nxn threads so use this to quickly fill the shared memory array
	#endif
	#if CULL_LOWER_TRIANGULAR_SUM  // only works if we throw away the entire block, otherwise results cannot be guaranteed correct due to the parallel sum
	if (bx < by)
	{
		return;
	}
	#endif
	__syncthreads();  // very important so that the entire shared mem array is assigned before any threads use it

	//p.w < 0 == padding element
	//m.x = aminoacid index
	//m.y = charge
	//m.z = vdw radius
	//m.w = crowder if == CROWDER_IDENTIFIER

	float4 rIp = residuePositions[bx*blockDim.x+tx];
	float4 rJp = residuePositions[by*blockDim.x+ty];

	if (rIp.w == rJp.w || rIp.w < CROWDER_IDENTIFIER || rJp.w < CROWDER_IDENTIFIER )		// same molecule || padding residues || padding residues
	{
		// residueIp.w == residueJp.w means that they are the same molecule
		// if either is -1 then its a padding residue and must be ignored
	}
	else
	{

		float4 rIm = residueMeta[bx*blockDim.x+tx];
		float4 rJm = residueMeta[by*blockDim.x+ty];

		float r = length(rIp,rJp) + EPS;  // add eps so that r is never 0, can happen in a collision


		//if there are crowders the LJ is replaced by the repulsive formula.
		float LJ;   // needs to be here for scoping issues
		float DH;

		#if REPULSIVE_CROWDING
		if (rIm.w == CROWDER_IDENTIFIER || rJm.w == CROWDER_IDENTIFIER) // repulsive crowder interaction
		{
			LJ = crowderPotential(r);
		}
		else  // normal LJ interaction
		{
		#endif
			// do the debye huckel long range intereactions
			DH = rIm.y * rJm.y * 1.602176487f * 1.602176487f * expf(-r/Xi) / r * DH_CONVERSION_FACTOR;
			//int ijX =  rint(rIm.x + float(AA_COUNT)*rJm.x);
			float sigmaij = (rIm.z + rJm.z ) * 0.5f;

		#if	LJ_LOOKUP_METHOD == SHARED_MEM
			float Eij = lambda*(sharedLJ[ijX] - e0);
		#elif LJ_LOOKUP_METHOD == TEXTURE_MEM
			float Eij = lambda*(tex2D(LJTexture,rIm.x,rJm.x) - e0);
		#elif LJ_LOOKUP_METHOD == CONST_MEM || LJ_LOOKUP_METHOD == GLOBAL_MEM
			float Eij = lambda*(LJPotentialData[ijX] - e0);
		#endif

			// sigmaij is the average atomic radius determined by the van der waals radius in kim2008
			//float r0 = sigmaij*1.122462048309372981433533049679f; //pow(2.0,(1.0/6.0));
			// the following 2 lines are needed for preserving numerical accuracy on a cpu. not needed on GPU
			float LJtmp = __powf(sigmaij/r, 6.0f);

			LJ = -4.0f*Eij*LJtmp*(LJtmp-1.0f);
			if (Eij>0.0f && r<sigmaij*1.122462048309372981433533049679f)
			{
				LJ = -LJ + 2.0f*Eij;
			}

		#if REPULSIVE_CROWDING
		}  // end conditional branch for LJ or repulsive short-range energy
		#endif
		//LJ = LJ * LJ_CONVERSION_FACTOR;  // convert from units used to K_bT

		// store the sum of all potentials for this thread so that we dont store to global mem until we have to.
#if CULL_LOWER_TRIANGULAR_SUM
		if (bx==by)
		{
			sharedmem_results[tx*blockDim.x+ty] = 0.5f*(LJ * LJ_CONVERSION_FACTOR + DH); // convert to kcal/mol
		}
		else // if (bx > by)
		{
			sharedmem_results[tx*blockDim.x+ty] = (LJ * LJ_CONVERSION_FACTOR + DH); // convert to kcal/mol
		}
#else
		sharedmem_results[tx*blockDim.x+ty] = 0.5f*(LJ * LJ_CONVERSION_FACTOR + DH) ; // convert to kcal/mol
#endif //  CULL_LOWER_TRIANGULAR_SUM
	}

	__syncthreads();  // all threads must have computed values so that they can be read concurrently from shared memory
	for(int bitMask=SHARED_MEM_RESULT_SIZE/2; bitMask>0; bitMask/=2)
	{
		float t=sharedmem_results[tx*blockDim.x+ty] + sharedmem_results[(tx*blockDim.x+ty)^bitMask];
		__syncthreads(); // make sure all values are read before modifying them
		sharedmem_results[tx*blockDim.x+ty]=t;
		__syncthreads(); // make sure all values are written back before the next iteration
	}

	// write results of the calculation back to global memory using a reduction summation

	result[bx*gridDim.x+by] = sharedmem_results[0];

	return;
};
*/
/*
__global__ void E_TiledKernelSMopt(float4 * residuePositions, float4 * residueMeta, int * residueCount, int * moleculePositions, int * moleculeCount, float* LJPotentialData, float* result)
{
	extern __shared__ float sharedmem_results[];	// array of temp results to sum, one per thread run.
	const int positionOfPosition = blockDim.x/4;
	extern __shared__ float4 X_tile_residuePositions[];
	const int positionOfMeta = __mul24(5,blockDim.x)/4;
	extern __shared__ float4 X_tile_residueMeta[];

	//__shared__ float sharedmem_results[TILE_DIM];	// array of temp results to sum, one per thread run.
	sharedmem_results[tx] = 0.0f;//EPS;
	#if  CULL_LOWER_TRIANGULAR_SUM  // only works if we throw away the entire block, otherwise results cannot be guaranteed correct due to the parallel sum
	if (bx < by)
		return;
	#endif

	#if POSITIONDATA_MEMORY == TEXTURE_MEM
		float4 yresiduep = tex1Dfetch(residuePositionTex, by*blockDim.x+tx);
	#else
		float4 yresiduep = residuePositions[by*blockDim.x+tx];
	#endif
	#if METADATA_MEMORY == TEXTURE_MEM
		float4 yresiduem = tex1Dfetch(residueMetaTex, by*blockDim.x+tx);
	#else
		float4 yresiduem = residueMeta[by*blockDim.x+tx];
	#endif
	// think of the position as a column of residues so 0 == by:0,tx:0, N = by:gridDimY*bDX+tx

	// copy residues from GRAM to shared memory tiles

	//__shared__ float4 X_tile_residuePositions[TILE_DIM];
	//__shared__ float4 X_tile_residueMeta[TILE_DIM];
	#if POSITIONDATA_MEMORY == SHARED_MEM
		X_tile_residuePositions[positionOfPosition + tx] = residuePositions[bx*blockDim.x+tx];
	#endif
	#if METADATA_MEMORY == SHARED_MEM
		X_tile_residueMeta[positionOfMeta + tx] = residueMeta[bx*blockDim.x+tx];
	#endif

	__syncthreads();  // very important so that the entire shared mem array is assigned before any threads use it

	if (yresiduep.w > PADDER_IDENTIFIER) // the residue in question is not a padding element
	{
		float c_lj(0.0f);   // Kahan summation variables
		float c_dh(0.0f);   // Kahan summation variables
		float lj_subtotal(0.0f);
		float dh_subtotal(0.0f);
		float y; // Kahan summation variables
		float t; // Kahan summation variables

		//#pragma unroll 4 - cant unroll dynamic variable
		for (int i = 0; i < blockDim.x; i++)
		{
			//#if POSITIONDATA_MEMORY == TEXTURE_MEM
			//	float4 POS = tex1Dfetch(residuePositionTex,(bx*blockDim.x+i));
			//#endif

			if (yresiduep.w == X_tile_residuePositions[positionOfPosition + i].w || X_tile_residuePositions[positionOfPosition + i].w < 0.0f )		// same molecule || padding residue
			{
				// Xpos.w == Ypos.w means that they are the same molecule
				// if either is -2 then its a padding residue and must be ignored
				// DO NOTHING
			}
			else
			{
				//#if METADATA_MEMORY == TEXTURE_MEM
				//float4 meta = tex1Dfetch(residueMetaTex,(bx*blockDim.x+i));
				//#endif

				//m.x = aminoacid index
				//m.y = charge
				//m.z = vdw radius
				//m.w = crowder if == CROWDER_IDENTIFIER
				float r(length(yresiduep,X_tile_residuePositions[positionOfPosition + i]) + EPS);  // add eps so that r is never 0, can happen in a collision

				float LJ(0.0f);
				float DH(0.0f);

				#if REPULSIVE_CROWDING
					//#if METADATA_MEMORY == TEXTURE_MEM
					//if (yresiduem.w == CROWDER_IDENTIFIER || meta.w == CROWDER_IDENTIFIER) // repulsive crowder interaction
					//#else
					if (yresiduem.w == CROWDER_IDENTIFIER || X_tile_residueMeta[positionOfMeta + i].w == CROWDER_IDENTIFIER) // repulsive crowder interaction
					//#endif
					{
						LJ = crowderPotential(r);//__powf(6.0f/r,6);
					}
					else  // normal LJ interaction
					{
				#endif

				//do the texture fetch first
				#if LJ_LOOKUP_METHOD == TEXTURE_MEM
					int ijX(rint( AA_COUNT*yresiduem.x + X_tile_residueMeta[positionOfMeta + i].x));
					float Eij(lambda*(tex1Dfetch(LJTexture,ijX) - e0));
					//float Eij(lambda*(e0 - e0));
					//float Eij(lambda*(tex2D(LJTexture2D,rint(X_tile_residueMeta[positionOfMeta + i].x),rint(yresiduem.x)) - e0));
				#else
					int ijX(rint( AA_COUNTf*X_tile_residueMeta[positionOfMeta + i].x + yresiduem.x));
					#if LJ_LOOKUP_METHOD == SHARED_MEM
					float Eij = lambda*(sharedLJ[ijX] - e0);
					#else  // __global__ or __constant__
					float Eij = lambda*(LJPotentialData[ijX] - e0);
					#endif
				#endif

				//#if METADATA_MEMORY == TEXTURE_MEM
				//	int ijX =  rint( yresiduem.x + AA_COUNTf*meta.x);

				//	DH = dhPotential(yresiduem.y,meta.y,r);
				//#else

					//old int ijX(rint( yresiduem.x + AA_COUNTf*X_tile_residueMeta[i].x));
					//old lookup for 1d tex
					//int ijX(rint( AA_COUNTf*X_tile_residueMeta[positionOfMeta + i].x + yresiduem.x));

					DH = dhPotential(yresiduem.y,X_tile_residueMeta[positionOfMeta + i].y,r);
				//#endif

				// compensate sum
				y = DH - c_dh;
				t = dh_subtotal + y;
				c_lj = (t-dh_subtotal)-y;
				dh_subtotal = t;




					// sigmaij is the average atomic radius determined by the van der waals radius in kim2008
//					float sigmaij = __fmul_rn(__fadd_rn(yresiduem.z,X_tile_residueMeta[i].z), 0.5f);
					//#if METADATA_MEMORY == TEXTURE_MEM
					//	float sigmaij((yresiduem.z + meta.z) * 0.5f);
					//#else
						float sigmaij((yresiduem.z + X_tile_residueMeta[positionOfMeta + i].z) * 0.5f);
					//#endif

					float LJtmp(__powf(sigmaij/r,6.0f));
					//float r0 = sigmaij*1.122462048309372981433533049679f; //sigma*pow(2.0,(1.0/6.0));

					LJ = -4.0f*Eij*LJtmp*(LJtmp-1.0f);
					//LJ = __fmul_rn(-4.0f,__fmul_rn(Eij,__fmul_rn(LJtmp,__fadd_rn(LJtmp,-1.0f))));
					if (Eij>0.0f && r < (sigmaij*1.122462048309372981433533049679f))
					{
						LJ = -LJ + 2.0f*Eij;
					}

				#if REPULSIVE_CROWDING
				}  // end conditional branch for LJ or repulsive short-range energy
				#endif
				//sharedmem_results[tx] += (LJ * LJ_CONVERSION_FACTOR) + (DH * DH_CONVERSION_FACTOR * 1.602176487f * 1.602176487f );
				y = LJ - c_lj;
				t = lj_subtotal + y;
				c_lj = (t-lj_subtotal)-y;
				lj_subtotal = t;


			} // if !(X_tile_residuePositions.w == Y_tile_residuePositions.w || X_tile_residuePositions.w < CROWDER_IDENTIFIER )
			sharedmem_results[tx] = (lj_subtotal * LJ_CONVERSION_FACTOR) + (dh_subtotal * DH_CONVERSION_FACTOR * 1.602176487f * 1.602176487f);
		} // for i = 0..Bdx

		#if CULL_LOWER_TRIANGULAR_SUM
		if (bx==by)
			sharedmem_results[tx] = sharedmem_results[tx]*0.5f;
		#else
		sharedmem_results[tx] *= 0.5f
		#endif //  CULL_LOWER_TRIANGULAR_SUM
	} // if !padding

	__syncthreads();  // all threads must have computed values so that they can be read concurrently from shared memory

	// do reduction, see SDK reduction example
	// FULLY unrolled reduction

	// do reduction in shared mem
    if (blockDim.x >= 512) { if (tx < 256) { sharedmem_results[tx] += sharedmem_results[tx + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tx < 128) { sharedmem_results[tx] += sharedmem_results[tx + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tx <  64) { sharedmem_results[tx] += sharedmem_results[tx + 64];  } __syncthreads(); }

	if (tx < 32)
	{
		if (blockDim.x >=  64) { sharedmem_results[tx] += sharedmem_results[tx + 32]; }
		if (blockDim.x >=  32) { sharedmem_results[tx] += sharedmem_results[tx + 16]; }
		if (blockDim.x >=  16) { sharedmem_results[tx] += sharedmem_results[tx +  8]; }
		if (blockDim.x >=   8) { sharedmem_results[tx] += sharedmem_results[tx +  4]; }
		if (blockDim.x >=   4) { sharedmem_results[tx] += sharedmem_results[tx +  2]; }
		if (blockDim.x >=   2) { sharedmem_results[tx] += sharedmem_results[tx +  1]; }
	}

	// write results of the calculation back to global memory, only thread 0 has the correct answer
	// only one warp processes < 32, therefore no syncthreads required here
	if (tx == 0)
		result[bx*gridDim.x+by] = sharedmem_results[tx];

	return;
};*/

///* full kernel with optional texture containers
__global__ void E_TiledKernel(float4 * residuePositions, float4 * residueMeta, int * residueCount, int * moleculePositions, int * moleculeCount, float* LJPotentialData, float* result)
{
	extern __shared__ float sharedmem_results[];	// array of temp results to sum, one per thread run.
	#if POSITIONDATA_MEMORY == SHARED_MEM
	const int positionOfPosition = blockDim.x/4;
	extern __shared__ float4 X_tile_residuePositions[];
	#endif
	#if METADATA_MEMORY == SHARED_MEM
	#if POSITIONDATA_MEMORY == SHARED_MEM
	const int positionOfMeta = __mul24(5,blockDim.x)/4;
	#else
	const int positionOfMeta = blockDim.x/4;
	#endif
	extern __shared__ float4 X_tile_residueMeta[];
	#endif

	//__shared__ float sharedmem_results[TILE_DIM];	// array of temp results to sum, one per thread run.
	sharedmem_results[tx] = 0.0f;//EPS;
	#if  CULL_LOWER_TRIANGULAR_SUM  // only works if we throw away the entire block, otherwise results cannot be guaranteed correct due to the parallel sum
	if (bx < by)
		return;
	#endif

	#if POSITIONDATA_MEMORY == TEXTURE_MEM
		float4 yresiduep = tex1Dfetch(residuePositionTex, by*blockDim.x+tx);
	#else
		float4 yresiduep = residuePositions[by*blockDim.x+tx];
	#endif
	#if METADATA_MEMORY == TEXTURE_MEM
		float4 yresiduem = tex1Dfetch(residueMetaTex, by*blockDim.x+tx);
	#else
		float4 yresiduem = residueMeta[by*blockDim.x+tx];
	#endif
	// think of the position as a column of residues so 0 == by:0,tx:0, N = by:gridDimY*bDX+tx

	// copy residues from GRAM to shared memory tiles

	//__shared__ float4 X_tile_residuePositions[TILE_DIM];
	//__shared__ float4 X_tile_residueMeta[TILE_DIM];
	#if POSITIONDATA_MEMORY == SHARED_MEM
		X_tile_residuePositions[positionOfPosition + tx] = residuePositions[bx*blockDim.x+tx];
	#endif
	#if METADATA_MEMORY == SHARED_MEM
		X_tile_residueMeta[positionOfMeta + tx] = residueMeta[bx*blockDim.x+tx];
	#endif

	__syncthreads();  // very important so that the entire shared mem array is assigned before any threads use it

	if (yresiduep.w > PADDER_IDENTIFIER) // the residue in question is not a padding element
	{
		float lj_subtotal(0.0f);
		float dh_subtotal(0.0f);
		#if COMPENSATE_KERNEL_SUM
		float c_lj(0.0f);   // Kahan summation variable
		float c_dh(0.0f);
		float y;
		float t;

		//		function kahanSum(input)
		//		 var potential = 0
		//		 var c = 0
		//		 for i = 0 to blockdim-1
		//		  y = lj+dh - c
		//		  t = potential + y
		//		  c = (t - potential) - y
		//		  potential = t
		//		 next i
		//		return sum

		#endif
		//#pragma unroll 4 - cant unroll dynamic variable
		for (int i = 0; i < blockDim.x; i++)
		{
			#if POSITIONDATA_MEMORY == TEXTURE_MEM
			float4 pos = tex1Dfetch(residuePositionTex,(bx*blockDim.x+i));
			if (yresiduep.w == pos.w || pos.w < 0.0f )		// same molecule || padding residue
			#else
			if (yresiduep.w == X_tile_residuePositions[positionOfPosition + i].w || X_tile_residuePositions[positionOfPosition + i].w < 0.0f )		// same molecule || padding residue
			#endif
			{
				// Xpos.w == Ypos.w means that they are the same molecule
				// if either is -2 then its a padding residue and must be ignored
				// DO NOTHING
			}
			else
			{
				#if METADATA_MEMORY == TEXTURE_MEM
				float4 meta = tex1Dfetch(residueMetaTex,(bx*blockDim.x+i));
				#endif
				//m.x = aminoacid index
				//m.y = charge
				//m.z = vdw radius
				//m.w = crowder if == CROWDER_IDENTIFIER
				#if POSITIONDATA_MEMORY == TEXTURE_MEM
				float r(length(yresiduep,pos) + EPS);  // add eps so that r is never 0, can happen in a collision
				#else
				float r(length(yresiduep,X_tile_residuePositions[positionOfPosition + i]) + EPS);  // add eps so that r is never 0, can happen in a collision
				#endif
				float LJ(0.0f);
				float DH(0.0f);

				#if REPULSIVE_CROWDING
					#if METADATA_MEMORY == TEXTURE_MEM
					if (yresiduem.w == CROWDER_IDENTIFIER || meta.w == CROWDER_IDENTIFIER) // repulsive crowder interaction
					#else
					if (yresiduem.w == CROWDER_IDENTIFIER || X_tile_residueMeta[positionOfMeta + i].w == CROWDER_IDENTIFIER) // repulsive crowder interaction
					#endif
					{
						LJ = crowderPotential(r);//__powf(6.0f/r,6);
					}
					else  // normal LJ interaction
					{
				#endif

				#if METADATA_MEMORY == TEXTURE_MEM
					int ijX(rint( AA_COUNT*yresiduem.x + meta.x));
				#else
					int ijX(rint( AA_COUNT*yresiduem.x + X_tile_residueMeta[positionOfMeta + i].x));
				#endif
				//do the texture fetch first
				#if LJ_LOOKUP_METHOD == TEXTURE_MEM
					float Eij(lambda*(tex1Dfetch(LJTexture,ijX) - e0));
					//float Eij(lambda*(e0 - e0));
				#elif LJ_LOOKUP_METHOD == SHARED_MEM
					float Eij(lambda*(sharedLJ[ijX] - e0));
				#else  // __global__ or __constant__
					float Eij(lambda*(LJPotentialData[ijX] - e0));
				#endif

				#if METADATA_MEMORY == TEXTURE_MEM
					DH = dhPotential(yresiduem.y,meta.y,r);
				#else
					DH = dhPotential(yresiduem.y,X_tile_residueMeta[positionOfMeta + i].y,r);
				#endif

				#if COMPENSATE_KERNEL_SUM
				// compensate sum

				y = DH - c_dh;
				t = dh_subtotal + y;
				c_lj = (t-dh_subtotal)-y;
				dh_subtotal = t;
				#else
				dh_subtotal += DH;
				#endif



					// sigmaij is the average atomic radius determined by the van der waals radius in kim2008
//					float sigmaij = __fmul_rn(__fadd_rn(yresiduem.z,X_tile_residueMeta[i].z), 0.5f);
					#if METADATA_MEMORY == TEXTURE_MEM
						float sigmaij((yresiduem.z + meta.z) * 0.5f);
					#else
						float sigmaij((yresiduem.z + X_tile_residueMeta[positionOfMeta + i].z) * 0.5f);
					#endif

					float LJtmp(__powf(sigmaij/r,6.0f));
					//float r0 = sigmaij*1.122462048309372981433533049679f; //sigma*pow(2.0,(1.0/6.0));

					LJ = -4.0f*Eij*LJtmp*(LJtmp-1.0f);
					//LJ = __fmul_rn(-4.0f,__fmul_rn(Eij,__fmul_rn(LJtmp,__fadd_rn(LJtmp,-1.0f))));
					if (Eij>0.0f && r < (sigmaij*1.122462048309372981433533049679f))
					{
						LJ = -LJ + 2.0f*Eij;
					}

				#if REPULSIVE_CROWDING
				}  // end conditional branch for LJ or repulsive short-range energy
				#endif
				//sharedmem_results[tx] += (LJ * LJ_CONVERSION_FACTOR) + (DH * DH_CONVERSION_FACTOR * 1.602176487f * 1.602176487f ); // moved outside loop

				#if COMPENSATE_KERNEL_SUM
				y = LJ - c_lj;
				t = lj_subtotal + y;
				c_lj = (t-lj_subtotal)-y;
				lj_subtotal = t;
				#else
				lj_subtotal += LJ;
				#endif

			} // if !(X_tile_residuePositions.w == Y_tile_residuePositions.w || X_tile_residuePositions.w < CROWDER_IDENTIFIER )
			sharedmem_results[tx] = (lj_subtotal * LJ_CONVERSION_FACTOR) + (dh_subtotal * DH_CONVERSION_FACTOR * 1.602176487f * 1.602176487f);
		} // for i = 0..Bdx

		#if CULL_LOWER_TRIANGULAR_SUM
		if (bx==by)
			sharedmem_results[tx] = sharedmem_results[tx]*0.5f;
		#else
		sharedmem_results[tx] *= 0.5f
		#endif //  CULL_LOWER_TRIANGULAR_SUM
	} // if !padding

	__syncthreads();  // all threads must have computed values so that they can be read concurrently from shared memory

	// do reduction, see SDK reduction example
	// FULLY unrolled reduction

	// do reduction in shared mem
    if (blockDim.x >= 512) { if (tx < 256) { sharedmem_results[tx] += sharedmem_results[tx + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tx < 128) { sharedmem_results[tx] += sharedmem_results[tx + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tx <  64) { sharedmem_results[tx] += sharedmem_results[tx + 64];  } __syncthreads(); }

	if (tx < 32)
	{
		if (blockDim.x >=  64) { sharedmem_results[tx] += sharedmem_results[tx + 32]; }
		if (blockDim.x >=  32) { sharedmem_results[tx] += sharedmem_results[tx + 16]; }
		if (blockDim.x >=  16) { sharedmem_results[tx] += sharedmem_results[tx +  8]; }
		if (blockDim.x >=   8) { sharedmem_results[tx] += sharedmem_results[tx +  4]; }
		if (blockDim.x >=   4) { sharedmem_results[tx] += sharedmem_results[tx +  2]; }
		if (blockDim.x >=   2) { sharedmem_results[tx] += sharedmem_results[tx +  1]; }
	}

	// write results of the calculation back to global memory, only thread 0 has the correct answer
	// only one warp processes < 32, therefore no syncthreads required here
	if (tx == 0)
		result[bx*gridDim.x+by] = sharedmem_results[tx];

	return;
};



/*__global__ void E_TiledLoopKernel(float4 * residuePositions, float4 * residueMeta, int * residueCount, int * moleculePositions, int * moleculeCount, float* LJPotentialData, float* result)
{
	// alternative method
	// we can also double buffer, will be better for small problem sizes
	__shared__ float sharedmem_results[TILE_DIM];	// array of temp results to sum, one per thread run.
	__shared__ float4 X_tile_residuePositions[TILE_DIM];
	__shared__ float4 X_tile_residueMeta[TILE_DIM];


	#if LJ_LOOKUP_METHOD == SHARED_MEM_LJ
		__shared__ float sharedLJ[AA_COUNT*AA_COUNT];
		for (int i=0;i<AA_COUNT*AA_COUNT;i+=AA_COUNT)
			memcpy(&(sharedLJ[i]),&(LJPotentialData[i]),sizeof(float)*AA_COUNT);
	#endif

	//__shared__ float sharedmem_results[TILE_DIM];	// array of temp results to sum, one per thread run.
	sharedmem_results[tx] = 0.0f;

	// save the local residue
	float4 yresiduep = residuePositions[bx*TILE_DIM+tx];
	float4 yresiduem = residueMeta[bx*TILE_DIM+tx];

	#define gx gridDim.x
	float mulfactor(0.5f);

	for (int address = bx*TILE_DIM; address <  TILE_DIM*gx; address+=TILE_DIM)
	{
		// copy residues from GRAM to shared memory tiles
		//__shared__ float4 X_tile_residuePositions[TILE_DIM];
		//__shared__ float4 X_tile_residueMeta[TILE_DIM];

		X_tile_residuePositions[tx] = residuePositions[address+tx];
		X_tile_residueMeta[tx]      = residueMeta[address+tx];

		// make sure no interactions are counted twice, when the preloaded tile address == block address * 1/2 to remove dupes
		// placed before the syncthreads because it does not rely on shared mem data

		__syncthreads();

		if (yresiduep.w > PADDER_IDENTIFIER) // the residue in question is not a padding element
		{
			for (unsigned int i = 0; i < TILE_DIM; i++)
			{
				if (yresiduep.w == X_tile_residuePositions[i].w || X_tile_residuePositions[i].w < 0.0f )		// same molecule || padding residue
				{
					// Xpos.w == Ypos.w means that they are the same molecule
					// if either is -2 then its a padding residue and must be ignored
					// DO NOTHING
				}
				else
				{
					//m.x = aminoacid index
					//m.y = charge
					//m.z = vdw radius
					//m.w = crowder if == CROWDER_IDENTIFIER

					float r = length(yresiduep,X_tile_residuePositions[i]) + EPS;  // add eps so that r is never 0, can happen in a collision
					float LJ = 0.0f;
					float DH = 0.0f;
					#if REPULSIVE_CROWDING
					if (yresiduem.w == CROWDER_IDENTIFIER || X_tile_residueMeta[i].w == CROWDER_IDENTIFIER) // repulsive crowder interaction
					{
						LJ = crowderPotential(r);//__powf(6.0f/r,6);
					}
					else  // normal LJ interaction
					{
					#endif


					//DH = yresiduem.y * X_tile_residueMeta[i].y  * __expf(-r/Xi) / r;
					DH = dhPotential(yresiduem.y,X_tile_residueMeta[i].y,r);


					int ijX =  rint( yresiduem.x + AA_COUNTf*X_tile_residueMeta[i].x);

					#if LJ_LOOKUP_METHOD == TEXTURE_MEM_LJ
						float Eij = lambda*(tex1Dfetch(LJTexture,ijX) - e0);
					#elif LJ_LOOKUP_METHOD == SHARED_MEM_LJ
						float Eij = lambda*(sharedLJ[ijX] - e0);
					#else  // __global__ or __constant__
						float Eij = lambda*(LJPotentialData[ijX] - e0);
					#endif


					// sigmaij is the average atomic radius determined by the van der waals radius in kim2008
					float sigmaij = (yresiduem.z + X_tile_residueMeta[i].z) * 0.5f;
					float LJtmp = __powf(sigmaij/r,6.0f);
					//float r0 = sigmaij*1.122462048309372981433533049679f; //sigma*pow(2.0,(1.0/6.0));
					//LJ = -4.0f*Eij*LJtmp*(LJtmp-1.0f);
					LJ = __fmul_rn(-4.0f,__fmul_rn(Eij,__fmul_rn(LJtmp,__fadd_rn(LJtmp,-1.0f))));
					if (Eij>0.0f && r < __fmul_rn(sigmaij,1.122462048309372981433533049679f))
					{
						LJ = -LJ + 2.0f*Eij;
					}

					#if REPULSIVE_CROWDING
					}  // end conditional branch for LJ or repulsive short-range energy
					#endif
					sharedmem_results[tx] += ((LJ * LJ_CONVERSION_FACTOR) + (DH * DH_CONVERSION_FACTOR * 1.602176487f * 1.602176487f ) )* mulfactor;
				} // if !(X_tile_residuePositions.w == Y_tile_residuePositions.w || X_tile_residuePositions.w < CROWDER_IDENTIFIER )
			} // for (unsigned int i = 0; i < TILE_DIM; i++)
		} // if (yresiduep.w > PADDER_IDENTIFIER) // the residue in question is not a padding element
		// sync to load next tile
		mulfactor = 1.0f;  // only the first set will be dupicated, so multiply by 1 each time now onwards
		__syncthreads();
	} // for (unsigned int address = bx*gx; address <  TILE_DIM*gx; address+=TILE_DIM)


	// do reduction, see SDK reduction example
    // FULLY unrolled reduction
    if (TILE_DIM >= 512) { if (tx < 256) { sharedmem_results[tx] += sharedmem_results[tx + 256]; } __syncthreads(); }
    if (TILE_DIM >= 256) { if (tx < 128) { sharedmem_results[tx] += sharedmem_results[tx + 128]; } __syncthreads(); }
    if (TILE_DIM >= 128) { if (tx <  64) { sharedmem_results[tx] += sharedmem_results[tx +  64]; } __syncthreads(); }
    if (tx < 32)
    {
        if (TILE_DIM >=  64) { sharedmem_results[tx] += sharedmem_results[tx + 32]; }
        if (TILE_DIM >=  32) { sharedmem_results[tx] += sharedmem_results[tx + 16]; }
        if (TILE_DIM >=  16) { sharedmem_results[tx] += sharedmem_results[tx +  8]; }
        if (TILE_DIM >=   8) { sharedmem_results[tx] += sharedmem_results[tx +  4]; }
        if (TILE_DIM >=   4) { sharedmem_results[tx] += sharedmem_results[tx +  2]; }
        if (TILE_DIM >=   2) { sharedmem_results[tx] += sharedmem_results[tx +  1]; }
    }

	// write results of the calculation back to global memory, only thread 0 has the correct answer
    if (tx == 0) result[bx] = sharedmem_results[tx];
	return;
};
*/

// performs a parallel sum on the data in values on the gpu with dimentions blockDim.x*blockDim.y
__global__ void parallelSum_kernel(float * values)
{
	#if REDUCTION_BLOCK_DIM > 256
		if (REDUCTION_BLOCK_DIM == 512) { if (tx < 256) { values[tx] += values[tx + 256]; } __syncthreads(); }
	#endif
	#if REDUCTION_BLOCK_DIM > 128
		if (REDUCTION_BLOCK_DIM >= 256) { if (tx < 128) { values[tx] += values[tx + 128]; } __syncthreads(); }
	#endif
	#if REDUCTION_BLOCK_DIM > 64
		if (REDUCTION_BLOCK_DIM >= 128) { if (tx <  64) { values[tx] += values[tx +  64]; } __syncthreads(); }
	#endif
		if (tx < 32)
		{
			if (REDUCTION_BLOCK_DIM >=  64) { values[tx] += values[tx + 32]; }
			if (REDUCTION_BLOCK_DIM >=  32) { values[tx] += values[tx + 16]; }
			if (REDUCTION_BLOCK_DIM >=  16) { values[tx] += values[tx +  8]; }
			if (REDUCTION_BLOCK_DIM >=   8) { values[tx] += values[tx +  4]; }
			if (REDUCTION_BLOCK_DIM >=   4) { values[tx] += values[tx +  2]; }
			if (REDUCTION_BLOCK_DIM >=   2) { values[tx] += values[tx +  1]; }
		}
};

__global__ void rotateMolecule_kernel (float4 *residuePositions, int *startPosition, int *length, float4* rotationVector, float4 *center)
{
	// cache frequently used variables
	__shared__ float4 centerV;
	__shared__ float4 q;

	centerV.x = center->x;
	centerV.y = center->y;
	centerV.z = center->z;

	// create rotation quaternion
	q.w = cos(rotationVector->w*0.5f);
	q.x = rotationVector->x*sin(rotationVector->w*0.5f);
	q.y = rotationVector->y*sin(rotationVector->w*0.5f);
	q.z = rotationVector->z*sin(rotationVector->w*0.5f);

	__syncthreads();

	// made it a unit quaternion
	float rq_len = rsqrt(q.x*q.x + q.y*q.y + q.z*q.z + q.w*q.w);

	__syncthreads();

	q.w = q.w*rq_len;
	q.x = q.x*rq_len;
	q.y = q.y*rq_len;
	q.z = q.z*rq_len;

	__syncthreads();

	//rotate the molecule using the quaternion made earlier

	#define vx (residuePositions[bx*blockDim.x+tx+startPosition[0]].x - centerV.x)
	#define vy (residuePositions[bx*blockDim.x+tx+startPosition[0]].y - centerV.y)
	#define vz (residuePositions[bx*blockDim.x+tx+startPosition[0]].z - centerV.z)

	if (bx*blockDim.x+tx < length[0])  // not addressing out of bounds
	{
		residuePositions[bx*blockDim.x+startPosition[0]+tx].x = ( vx*(q.w*q.w+q.x*q.x-q.y*q.y-q.z*q.z) + vy*(q.x*q.y-q.w*q.z)*2.0f  		+ vz*(q.w*q.y+q.x*q.z)*2.0f )        	 + centerV.x;
		residuePositions[bx*blockDim.x+startPosition[0]+tx].y = ( vx*(q.w*q.z+q.x*q.y)*2.0f   		 + vy*(q.w*q.w-q.x*q.x+q.y*q.y-q.z*q.z) + vz*(q.y*q.z-q.w*q.x)*2.0f ) 		     + centerV.y;
		residuePositions[bx*blockDim.x+startPosition[0]+tx].z = ( vx*(q.x*q.z-q.w*q.y)*2.0f   		 + vy*(q.y*q.z+q.w*q.x)*2.0f   		    + vz*(q.w*q.w-q.x*q.x-q.y*q.y+q.z*q.z) ) + centerV.z;
	}

};

__global__ void translateMolecule_kernel (float4 *residuePositions, int *startPosition, int *length, float4* translationVector, float4 *center)
{
	if (bx*blockDim.x+tx < length[0])  // not addressing out of bounds
	{
		residuePositions[bx*blockDim.x+startPosition[0]+tx].x = (residuePositions[bx*blockDim.x+startPosition[0]+tx].x + translationVector->x);
		residuePositions[bx*blockDim.x+startPosition[0]+tx].y = (residuePositions[bx*blockDim.x+startPosition[0]+tx].y + translationVector->y);
		residuePositions[bx*blockDim.x+startPosition[0]+tx].z = (residuePositions[bx*blockDim.x+startPosition[0]+tx].z + translationVector->z);
	}

	float3 tmpCenter;
	tmpCenter.x = center->x + translationVector->x;
	tmpCenter.y = center->y + translationVector->y;
	tmpCenter.z = center->z + translationVector->z;

	__syncthreads();

	center->x = tmpCenter.x;
	center->y = tmpCenter.y;
	center->z = tmpCenter.z;
};




#endif //USING CUDA
#endif
